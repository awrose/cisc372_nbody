#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include "vector.h"
#include "config.h"

__global__ void computePairwiseAccels(vector3** accels, vector3* hPos, double* mass){
	int i = (blockDim.y * blockIdx.y) + threadIdx.y;
	int j = (blockDim.x * blockIdx.x) + threadIdx.x;

	if(i < NUMENTITIES && j < NUMENTITIES){
		if(i == j){
			FILL_VECTOR(accels[i][j],0,0,0);
		}else{
			vector3 distance;
			for (int k=0;k<3;k++) distance[k]=hPos[i][k]-hPos[j][k];
			double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
			double magnitude=sqrt(magnitude_sq);
			double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
			FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);			
		}
	}
}

__global__ void computeSum(vector3** accels, vector3* hPos, vector3* hVel){
	int in = blockIdx.x;
	if(in < NUMENTITIES){
		vector3 accel_sum={0,0,0};
		for (int j=0;j<NUMENTITIES;j++){
			for (int k=0;k<3;k++)
				//printf("screaming crying throwing up");
				accel_sum[k]+=accels[in][j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (int k=0;k<3;k++){
			hVel[in][k]+=accel_sum[k]*INTERVAL;
			hPos[in][k]+=hVel[in][k]*INTERVAL;
		}
	}

}

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){

	dim3 block_size(16,16);
	dim3 block_count((NUMENTITIES+15) / block_size.x, (NUMENTITIES+15) / block_size.y);
	computePairwiseAccels<<<block_count, block_size>>>(accels, d_hPos, d_mass);
	hipDeviceSynchronize();

	dim3 grid_dims (NUMENTITIES, 1, 1);
	computeSum<<<grid_dims, 3>>>(accels, d_hPos, d_hVel);
	hipDeviceSynchronize();

}